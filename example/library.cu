#include "hip/hip_runtime.h"
#include "library.cuh"

__global__ void add(double *a, double *b, size_t len) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        a[i] = a[i] + b[i];
    }
}

extern "C" {
int add_arr(double *a, double *b, size_t len) {
    double *dev_a, *dev_b;
    size_t arr_size_bytes = len * sizeof(double),
            block_size = 1024,
            num_blocks = (size_t) std::ceil(((double) len) / ((double) block_size));
    hipError_t err = hipMalloc((void **) &dev_a, arr_size_bytes);
    if (err != hipSuccess) {
        goto cleanup;
    }

    err = hipMalloc((void **) &dev_b, arr_size_bytes);
    if (err != hipSuccess) {
        goto cleanup;
    }

    err = hipMemcpy(dev_a, a, arr_size_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        goto cleanup;
    }

    err = hipMemcpy(dev_b, b, arr_size_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        goto cleanup;
    }

    add<<<num_blocks, block_size>>>(dev_a, dev_b, len);

    err = hipMemcpy(a, dev_a, arr_size_bytes, hipMemcpyDeviceToHost);

    cleanup:
    hipError_t cleanup_err = hipSuccess;
    if (dev_a != nullptr) {
        cleanup_err = hipFree(dev_a);
    }
    if (dev_b != nullptr) {
        hipError_t err_b = hipFree(dev_b);
        if (cleanup_err == hipSuccess) {
            cleanup_err = err_b;
        }
    }
    if (err == hipSuccess) {
        err = cleanup_err;
    }
    return err;
}
}